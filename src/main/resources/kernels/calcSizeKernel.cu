#include "hip/hip_runtime.h"
__kernel void sizeKernel(__global const int *field,
                           const uint size,
                           __global int *result)
{
    int gid = get_global_id(0);
    int field_size = 300;

    int x_max = min((uint) (gid % field_size) + size, (uint) 300);
    int y_max = min((uint) (gid / field_size) + size, (uint) 300);
    int sum = 0;

    for (int x = gid % field_size; x < x_max; x++)
    {
        for (int y = gid / field_size; y < y_max; y++)
        {
            sum += field[x + y * field_size];
        }
    }

    result[gid] = sum;
}