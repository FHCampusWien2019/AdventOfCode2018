#include "hip/hip_runtime.h"
__kernel void sizeKernel(__global const int *field,
                           const uint size,
                           __global int *result)
{
    int gid = get_group_id(0);
    int lid = get_local_id(0);
    int field_size = 300;
    int x = lid + (150 * (gid & 1));
    int y = gid / 2;

    int x_max = min((uint) x + size, (uint) 300);
    int y_max = min((uint) y + size, (uint) 300);
    int sum = 0;

    for (int i = y; i < y_max; i++)
    {
        for (int j = x; j < x_max; j++)
        {
            sum += field[j + i * field_size];
        }
    }

    result[x + y * field_size] = sum;
}